
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <cctype>

//__device__ std::string d_content;
//__device__ int d_result[26];
//__device__ int d_content_length;

int main()
{
	std::string abcString = "abcdefghijklmnopqrstuvwxyz";
	int h_result[26];

	for (int i = 0; i < 26; i++)
	{
		h_result[i] = 0;
	}

	unsigned int content_length;
	std::ifstream ifs("text.txt");
	std::string content((std::istreambuf_iterator<char>(ifs)), (std::istreambuf_iterator<char>()));
	std::transform(content.begin(), content.end(), content.begin(), [](unsigned char c) { return std::tolower(c); });
	
	content_length = content.size();

	//CPU solution
	for (int i = 0; i < content_length; i++)
	{
		if ('a' <= content[i] && content[i] <= 'z')
		{
			for (int j = 0; j < 26; j++)
			{
				if (content[i] == abcString[j])
				{
					h_result[j]++;
				}
			}
		}
	}

	std::cout << "CPU result:" << std::endl;
	for (int i = 0; i < 26; i++)
	{
		std::cout << abcString[i] << ": " << h_result[i] << std::endl;
	}

	std::cin.ignore();
	return 0;
}

//__global__ int gpu_solution() {
//	return 0;
//}
