
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <cctype>

//__device__ std::string d_content;
//__device__ int d_result[26];
//__device__ int d_content_length;

int content_length;

int main()
{
	std::string abcString = "abcdefghijklmnopqrstuvwxyz";
	int h_result[26];

	for (int i = 0; i < 26; i++)
	{
		h_result[i] = 0;
	}
	
	std::ifstream ifs("text.txt");
	std::string content((std::istreambuf_iterator<char>(ifs)), (std::istreambuf_iterator<char>()));
	std::transform(content.begin(), content.end(), content.begin(), [](unsigned char c) { return std::tolower(c); });
	
	content_length = content.size();

	//CPU solution
	for (int i = 0; i < content_length; i++)
	{
		if ('a' <= content[i] && content[i] <= 'z')
		{
			for (int j = 0; j < 26; j++)
			{
				if (content[i] == abcString[j])
				{
					h_result[j]++;
				}
			}
		}
	}

	std::cout << "CPU result:" << std::endl;
	for (int i = 0; i < 26; i++)
	{
		std::cout << abcString[i] << ": " << h_result[i] << std::endl;
	}

	std::cin.ignore();
	return 0;
}

__device__ char d_content[8000000];
__device__ char d_abc[26];
__device__ int d_result[26];

__global__ void gpu_solution() {
	int i = threadIdx.x * blockIdx.x;

	if ('a' <= d_content[i] && d_content[i] <= 'z')
	{
		for (int j = 0; j < 26; j++)
		{
			if (d_abc[j] == d_content[i])
			{
				d_result[j]++;
			}
		}
	}
}
