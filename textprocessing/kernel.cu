
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>

int main()
{
	std::ifstream ifs("text.txt");
	std::string content((std::istreambuf_iterator<char>(ifs)), (std::istreambuf_iterator<char>()));

	std::cout << content;

	std::cin.ignore();
	return 0;
}
