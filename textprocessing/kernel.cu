
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <cctype>
#include <ctime>

#define N 8000000
#define M 26

//__device__ std::string d_content;
//__device__ int d_result[26];
//__device__ int d_content_length;

int content_length;

__device__ char d_content[N];
__device__ char d_abc[M];
__device__ int d_result[M];

__global__ void gpu_solution() {
	int i = threadIdx.x * blockIdx.x;

	if ('a' <= d_content[i] && d_content[i] <= 'z')
	{
		for (int j = 0; j < 26; j++)
		{
			if (d_abc[j] == d_content[i])
			{
				d_result[j]++;
			}
		}
	}
}

int main()
{
	std::string abcString = "abcdefghijklmnopqrstuvwxyz";
	int h_result[M];

	for (int i = 0; i < M; i++)
	{
		h_result[i] = 0;
	}
	
	std::ifstream ifs("text.txt");
	std::string content((std::istreambuf_iterator<char>(ifs)), (std::istreambuf_iterator<char>()));
	std::transform(content.begin(), content.end(), content.begin(), [](unsigned char c) { return std::tolower(c); });
	
	content_length = content.size();

	clock_t begin = clock();

	//CPU solution
	for (int i = 0; i < content_length; i++)
	{
		if ('a' <= content[i] && content[i] <= 'z')
		{
			for (int j = 0; j < M; j++)
			{
				if (content[i] == abcString[j])
				{
					h_result[j]++;
				}
			}
		}
	}

	clock_t end = clock();
	double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

	std::cout << "CPU result: " << elapsed_secs << std::endl;
	for (int i = 0; i < 26; i++)
	{
		std::cout << abcString[i] << ": " << h_result[i] << std::endl;
	}

	//h_result[M];

	//GPU solution

	begin = clock();

	hipMemcpyToSymbol(HIP_SYMBOL(d_content), &content, N, sizeof(char) * N);
	hipMemcpyToSymbol(HIP_SYMBOL(d_abc), &abcString, M, sizeof(char) * M);
	hipMemcpyToSymbol(HIP_SYMBOL(d_result), h_result, M, sizeof(int) * M);

	int block_size = N / 512;
	int temp = block_size * 512;

	if (N - temp != 0)
	{
		block_size++;
	}

	gpu_solution <<<block_size, 512 >>> ();

	hipMemcpyFromSymbol(&content, HIP_SYMBOL(d_content), N, sizeof(char) * N);
	hipMemcpyFromSymbol(&abcString, HIP_SYMBOL(d_abc), M, sizeof(char) * M);
	hipMemcpyFromSymbol(h_result, HIP_SYMBOL(d_result), M, sizeof(int) * M);
	hipDeviceSynchronize();

	end = clock();
	elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;

	std::cout << "GPU result: " << elapsed_secs << std::endl;
	for (int i = 0; i < 26; i++)
	{
		std::cout << abcString[i] << ": " << h_result[i] << std::endl;
	}

	std::cin.ignore();
	return 0;
}
